#include "hip/hip_runtime.h"
//
// Tomás Oliveira e Silva,  October 2024
//
// Arquiteturas de Alto Desempenho 2024/2025
//
// MD5 hash CUDA kernel code
//
// md5_cuda_kernel() --- each thread computes the MD5 hash of one message
//
// do not use this directy to search for DETI coins!
//

//
// needed stuff
//

typedef unsigned int u32_t;

#include "md5.h"
#define next_value_to_try(v)                              \
	do                                                      \
{                                                       \
	v++;                                                \
	if ( (v & 0xFFu) == 0x7Fu )                           \
	{                                                   \
		v += 0xA1u;               \
		if (((v >> 8) & 0xFFu ) == 0x7Fu)                 \
		{                                               \
			v += 0xA1u << 8;                             \
			if (((v >> 16) & 0xFFu ) == 0x7Fu)            \
			{                                           \
				v += 0xA1u << 16;                        \
				if (((v >> 24) & 0xFFu ) == 0x7Fu)        \
				v += 0xA1u << 24;                    \
			}                                           \
		}                                               \
	}                                                   \
} while(0)
// the nvcc compiler stores x[] and state[] in registers (constant indices!)
//
// global thread number: n = threadIdx.x + blockDim.x * blockIdx.x
// global warp number: n >> 5
// warp thread number: n & 31
//

extern "C" __global__ __launch_bounds__(128,1) void cuda_md5_kernel(u32_t v1,u32_t v2,u32_t *data_storage_device)
{
	u32_t number_coins=64u;	
	u32_t n,a,b,c,d,state[4],x[16],coin[13u],hash[4u];
	n = (u32_t)threadIdx.x + (u32_t)blockDim.x * (u32_t)blockIdx.x;
	//printf("Thread ID:%\n",n);
	u32_t previous;
	//
	// mandatory for a DETI coin
	//

	coin[ 0u] = 0x49544544; // ITED
	coin[ 1u] = 0x696f6320; // ioc_ 
	coin[ 2u] = 0x7343206e; // sC_n
	coin[ 3u] = 0x30324341; // 02CA
	coin[ 4u] = 0x41203432; // A_42
	coin[ 5u] = 0x34314441; // 41DA
	coin[ 6u] = v1; 
	coin[ 7u] = v2; 
	// insert some complexity with the thread ID 
	coin[ 8u] = 0x20202020; 
	coin[ 8u] += (n%64u)<< 0; n/=64u;
	coin[ 8u] += (n%64u)<< 8; n/=64u;
	coin[ 8u] += (n%64u)<< 16; n/=64u;
	coin[ 8u] += (n%64u)<< 24; n/=64u;
	coin[ 9u] = 0x20202020; 
	coin[ 10u] = 0x20202020; 
	coin[ 11u] = 0x20202020; 
	coin[ 12u] = 0x0A202020; 

	for (n=0u;n< number_coins;n++){

# define C(c)         (c)
# define ROTATE(x,n)  (((x) << (n)) | ((x) >> (32 - (n))))
# define DATA(idx)    coin[idx] 
# define HASH(idx)    hash[idx]
# define STATE(idx)   state[idx]
# define X(idx)       x[idx]
		CUSTOM_MD5_CODE();
# undef C
# undef ROTATE
# undef DATA
# undef HASH
# undef STATE
# undef X
		//gets the current idx if it has encoutered a new coin then increments it with an atomic add
		u32_t idx = data_storage_device[0];
		//printf("Coin: %52.52s",(char *)coin);
		if (hash[3] == 0 && idx < 1024 - 13) {
			idx = atomicAdd(&data_storage_device[0], 13); // Atomic index increment
			printf("Coin: %52.52s",(char *)coin);
			for (int j = 0; j < 13; j++) {
				// just to print the founded coin
				data_storage_device[idx + j] = coin[j]; // Copy coin to storage
			}
		}
		for(u32_t offset=9u;offset<13u;offset++){
			// change the values 
			previous=coin[offset-1u];
			if (previous == 0x7F7F7F7Fu)
			{
				next_value_to_try(coin[offset]);
			}

			next_value_to_try(coin[offset]);
		}
	}

}
